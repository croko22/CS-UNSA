#ifndef MLP_CUDA_H
#define MLP_CUDA_H

#include <vector>
#include <cmath>
#include <random>
#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(x)                                                                          \
    {                                                                                          \
        hipError_t err = x;                                                                   \
        if (err != hipSuccess)                                                                \
            std::cerr << #x << " failed with error: " << hipGetErrorString(err) << std::endl; \
    }

// Funciones auxiliares
void randomInit(double *data, int size)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-0.5, 0.5);
    for (int i = 0; i < size; ++i)
        data[i] = dis(gen);
}

// Kernel de suma ponderada
__global__ void computeWeightedSumKernel(
    double *weights,
    double *biases,
    double *input,
    double *output,
    int inSize,
    int outSize)
{
    int neuron = threadIdx.x + blockIdx.x * blockDim.x;
    if (neuron < outSize)
    {
        double sum = biases[neuron];
        for (int i = 0; i < inSize; ++i)
            sum += weights[neuron * inSize + i] * input[i];
        output[neuron] = sum;
    }
}

// Función de activación
__device__ double sigmoid(double x)
{
    return 1.0 / (1.0 + exp(-x));
}

__device__ double sigmoidDerivative(double x)
{
    return x * (1.0 - x);
}

__global__ void activateKernel(double *input, double *output, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
        output[idx] = sigmoid(input[idx]);
}

__global__ void derivativeActivateKernel(double *input, double *output, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
        output[idx] = sigmoidDerivative(input[idx]);
}

// Actualización de gradientes
__global__ void updateWeightsKernel(
    double *weights,
    double *deltas,
    double *inputs,
    int inSize,
    int outSize,
    double learningRate)
{
    int neuron = threadIdx.x + blockIdx.x * blockDim.x;
    if (neuron < outSize)
    {
        for (int i = 0; i < inSize; ++i)
            weights[neuron * inSize + i] += learningRate * deltas[neuron] * inputs[i];
    }
}

__global__ void updateBiasesKernel(double *biases, double *deltas, int size, double learningRate)
{
    int neuron = threadIdx.x + blockIdx.x * blockDim.x;
    if (neuron < size)
        biases[neuron] += learningRate * deltas[neuron];
}

struct Layer
{
    double *weights, *biases;     // Host
    double *d_weights, *d_biases; // Device
    int input_size, output_size;

    Layer(int in, int out) : input_size(in), output_size(out)
    {
        weights = new double[in * out];
        biases = new double[out];
        randomInit(weights, in * out);
        randomInit(biases, out);

        CHECK_CUDA(hipMalloc(&d_weights, in * out * sizeof(double)));
        CHECK_CUDA(hipMalloc(&d_biases, out * sizeof(double)));

        CHECK_CUDA(hipMemcpy(d_weights, weights, in * out * sizeof(double), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_biases, biases, out * sizeof(double), hipMemcpyHostToDevice));
    }

    ~Layer()
    {
        delete[] weights;
        delete[] biases;
        hipFree(d_weights);
        hipFree(d_biases);
    }
};

class MLP_CUDA
{
private:
    std::vector<Layer> layers;
    double *d_input, *d_output, *d_weightedSum;
    double *input_h, *output_h;

public:
    MLP_CUDA(const std::vector<int> &layerSizes)
    {
        for (size_t i = 1; i < layerSizes.size(); ++i)
        {
            layers.emplace_back(layerSizes[i - 1], layerSizes[i]);
        }
    }

    ~MLP_CUDA()
    {
        delete[] input_h;
        delete[] output_h;
        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_weightedSum);
    }

    std::vector<double> predict(const std::vector<double> &input)
    {
        int inputSize = input.size();
        input_h = new double[inputSize];
        for (int i = 0; i < inputSize; ++i)
            input_h[i] = input[i];

        CHECK_CUDA(hipMalloc(&d_input, inputSize * sizeof(double)));
        CHECK_CUDA(hipMemcpy(d_input, input_h, inputSize * sizeof(double), hipMemcpyHostToDevice));

        double *currentInput = d_input;
        int currentSize = inputSize;

        for (auto &layer : layers)
        {
            dim3 threads(256);
            dim3 blocks((layer.output_size + threads.x - 1) / threads.x);

            CHECK_CUDA(hipMalloc(&d_weightedSum, layer.output_size * sizeof(double)));
            CHECK_CUDA(hipMalloc(&d_output, layer.output_size * sizeof(double)));

            computeWeightedSumKernel<<<blocks, threads>>>(
                layer.d_weights, layer.d_biases, currentInput, d_weightedSum, currentSize, layer.output_size);
            activateKernel<<<blocks, threads>>>(d_weightedSum, d_output, layer.output_size);
            hipDeviceSynchronize();

            if (currentInput != d_input)
                hipFree(currentInput);
            currentInput = d_output;
            currentSize = layer.output_size;
        }

        output_h = new double[currentSize];
        CHECK_CUDA(hipMemcpy(output_h, currentInput, currentSize * sizeof(double), hipMemcpyDeviceToHost));

        std::vector<double> result(output_h, output_h + currentSize);
        delete[] output_h;

        return result;
    }

    void train(
        const std::vector<std::vector<double>> &X_train,
        const std::vector<std::vector<double>> &y_train,
        double learning_rate,
        int epochs)
    {
        int inputSize = X_train[0].size();
        int outputSize = y_train[0].size();

        double *d_error = nullptr;
        double *d_delta = nullptr;
        double *h_delta = new double[outputSize];

        for (int epoch = 0; epoch < epochs; ++epoch)
        {
            for (size_t sample = 0; sample < X_train.size(); ++sample)
            {
                auto prediction = predict(X_train[sample]);

                // Calcular error
                std::vector<double> target = y_train[sample];
                std::vector<double> errors(outputSize);
                for (int i = 0; i < outputSize; ++i)
                    errors[i] = target[i] - prediction[i];

                // Copiar a dispositivo
                CHECK_CUDA(hipMalloc(&d_error, outputSize * sizeof(double)));
                CHECK_CUDA(hipMemcpy(d_error, errors.data(), outputSize * sizeof(double), hipMemcpyHostToDevice));

                // Derivada de salida
                CHECK_CUDA(hipMalloc(&d_delta, outputSize * sizeof(double)));
                derivativeActivateKernel<<<dim3((outputSize + 255) / 256), dim3(256)>>>(d_output, d_delta, outputSize);
                hipDeviceSynchronize();

                for (int i = 0; i < outputSize; ++i)
                    h_delta[i] = errors[i] * (prediction[i] * (1.0 - prediction[i]));

                CHECK_CUDA(hipMemcpy(d_delta, h_delta, outputSize * sizeof(double), hipMemcpyHostToDevice));

                // Actualizar última capa
                Layer &lastLayer = layers.back();
                CHECK_CUDA(hipMemcpy(lastLayer.d_biases, h_delta, outputSize * sizeof(double), hipMemcpyHostToDevice));

                updateWeightsKernel<<<dim3((lastLayer.input_size * lastLayer.output_size + 255) / 256), dim3(256)>>>(
                    lastLayer.d_weights, d_delta, d_input, lastLayer.input_size, lastLayer.output_size, learning_rate);
                hipDeviceSynchronize();

                hipFree(d_error);
                hipFree(d_delta);
            }
        }

        delete[] h_delta;
    }
};

#endif // MLP_CUDA_H